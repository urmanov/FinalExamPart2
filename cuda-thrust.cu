#include "hip/hip_runtime.h"
#include <thrust/random.h>
#include <thrust/count.h>
#include <iostream>

const int N = 1000000; // Number of Monte-Carlo simulations.

struct random_point {
private:
	thrust::default_random_engine rng;
public:
	__device__
	float2 operator()(int index) {
		rng.discard(2*index);
		return make_float2(
			(float)rng() / thrust::default_random_engine::max,
			(float)rng() / thrust::default_random_engine::max);
	}
};

struct inside_circle {
private:
	__device__
	unsigned int inside(float2 p) const {
		return (((p.x-0.5)*(p.x-0.5)+(p.y-0.5)*(p.y-0.5))<0.25) ? 1 : 0;
	}
public:
	// Used for-on-the fly.
	__device__
	unsigned int operator()(int index) const {
		// Generate a random point.
		random_point point;
		return inside(point(index));
	}
};

int main()
{
  // DEVICE:
  thrust::counting_iterator<int> index(0);
  size_t total = thrust::count_if(index, index+N, inside_circle());

  // HOST: Print estimate of PI.
  std::cout << "PI: " << 4.0*(float)total/(float)N << std::endl;

  return 0;
}
